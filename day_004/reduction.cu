#include "hip/hip_runtime.h"
#define BLOCK_DIM 1024
#include <stdio.h>
#include <stdlib.h>

__global__ void reduce_kernel(float *input_d,float* partialSums, unsigned int N) {
  unsigned int segment = blockDim.x * blockIdx.x *2;
  unsigned int i = segment + threadIdx.x *2;
  for(int stride = 1; stride <=BLOCK_DIM ; stride *=2){
    if(threadIdx.x % stride ==0){
      input_d[i] += input_d[i+stride]
    }
    __syncthreads();
  }
  if(threadIdx.x==0){
    partialSums_d[..] = input_d[i]
  }
  
}


int main(float *input, int N) {
  // allocate memory
  hipEvent_t start, stop;
  float milliseconds = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  float *input_d;
  hipMalloc((**void) &input_d, sizeof(float));

  // copy data to device
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipMemcpy(input_d,input,N*sizeof(float),hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  printf("Time taken for hipMemcpy: %f ms\n", milliseconds);
  // Destroy events
  hipEventDestroy(start);
  hipEventDestroy(stop);


  // kernel config
  const unsigned int numThreadsPerBlock = BLOCK_DIM;
  const unsigned int numElementsPerBlock = numThreadsPerBlock*2;
  const unsigned int numBlocks = (N+numThreadsPerBlock-1)/numThreadsPerBlock;

  // allocate partial sums
  float* partialSums = (float*)hipMalloc(BLOCK_DIM*sizeof(float));
  float *partialSums_d;

  // call kernel
  reduce_kernel<<<numBlocks,numThreadsPerBlock>>>(input_d, partialSums_d, N);
  hipDeviceSynchronize();

  // copy data from gpu
  hipMemcpy(partialSums,partialSums_d,BLOCK_DIM*sizeof(float),hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // Reduce parial sums on CPU
  float sum = 0.0f;
  for(int i = 0; i<BLOCK_DIM;++i){
    sum += partialSums[i];
  }
  printf(sum)

  // free memory
  hipFree(input_d);
  hipFree(partialSums_d);

}