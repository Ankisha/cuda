#define BLOCK_DIM 1024

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void reduce_kernel(float *input_d,float* partialSums_d, unsigned int N) {
  unsigned int segment = blockDim.x * blockIdx.x *2;
  unsigned int i = segment + threadIdx.x ;
  for(int stride = BLOCK_DIM; stride >0 ; stride /=2){
    if(threadIdx.x<stride){
      input_d[i] += input_d[i+stride];
    }
    __syncthreads();
  }
  if(threadIdx.x==0){
    partialSums_d[blockIdx.x] = input_d[i];
  }
  
}


int main() {
  printf("%s","hello");
  float *h_input;
  int N = 32;
  // Initialize input array
  for (int i = 0; i < N; i++) {
      h_input[i] = 1.0f; // Set all to 1.0 for easy verification
  }
  // allocate memory
  hipEvent_t start, stop;
  float milliseconds = 0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  float *input_d;
  hipMalloc((void**) &input_d, N*sizeof(float));

  // copy data to device
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  hipMemcpy(input_d,h_input,N*sizeof(float),hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  printf("Time taken for cudaMemcpy: %f ms\n", milliseconds);
  // Destroy events
  hipEventDestroy(start);
  hipEventDestroy(stop);


  // kernel config
  const unsigned int numThreadsPerBlock = BLOCK_DIM;
  const unsigned int numElementsPerBlock = numThreadsPerBlock*2;
  const unsigned int numBlocks = (N+numThreadsPerBlock-1)/numThreadsPerBlock;

  // allocate partial sums
  float* partialSums = (float*)malloc(numBlocks*sizeof(float));
  float *partialSums_d;
  hipMalloc((void**)&partialSums_d,numBlocks*sizeof(float));
  hipDeviceSynchronize();

  // call kernel
  reduce_kernel<<<numBlocks,numThreadsPerBlock>>>(input_d, partialSums_d, N);
  hipDeviceSynchronize();

  // copy data from gpu
  hipMemcpy(partialSums,partialSums_d,BLOCK_DIM*sizeof(float),hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // Reduce parial sums on CPU
  float sum = 0.0f;
  for(int i = 0; i<BLOCK_DIM;++i){
    sum += partialSums[i];
  }
  printf("%f\n", sum);

  // free memory
  hipFree(input_d);
  hipFree(partialSums_d);

}